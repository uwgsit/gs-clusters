// Loosely based on SC11 nVidia CUDA tutorial - https://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <errno.h>
#include <string.h>
#include <math.h>

void usage();
__global__ void vector_add(double *,double *,double *,const unsigned long int);

int main(int argc,char **argv) {
    int c,device,device_count;
    unsigned long int i,blocks,threads,n;
    double *vecA,*vecB,*vecC,*d_vecA,*d_vecB,*d_vecC,sum;
    hipError_t cuda_error;

    n = 0;
    blocks = threads = 1;

    hipGetDevice(&device);
    hipGetDeviceCount(&device_count);

    fprintf(stderr,"CUDA device = %d/%d\n",device,device_count);

    while((c = getopt(argc,argv,"b:n:t:")) != -1) {
        switch(c) {
            case 'b':
                errno = 0;
                blocks = (unsigned long int)strtoul(optarg, (char**)NULL, 10);
                if(errno) {
                    fprintf(stderr,"Invalid input %s: %s\n",optarg,strerror(errno));
                    usage();
                    exit(EXIT_FAILURE);
                }
                break;
            case 'n':
                errno = 0;
                n = (unsigned long int)strtoul(optarg, (char**)NULL, 10);
                if(errno) {
                    fprintf(stderr,"Invalid input %s: %s\n",optarg,strerror(errno));
                    usage();
                    exit(EXIT_FAILURE);
                }
                break;
            case 't':
                errno = 0;
                threads = (unsigned long int)strtoul(optarg, (char**)NULL, 10);
                if(errno) {
                    fprintf(stderr,"Invalid input %s: %s\n",optarg,strerror(errno));
                    usage();
                    exit(EXIT_FAILURE);
                }
                break;
            case '?':
                usage();
                exit(EXIT_FAILURE);
                break;
            default:
                usage();
                abort();
                break;
        }
    }

    if(n <= 0 || blocks <= 0 || threads <= 0) {
        fprintf(stderr,"Supply all arguments!\n");
        usage();
        exit(EXIT_FAILURE);
    }

    if((vecA = (double *)malloc(sizeof(double)*n)) == NULL) {
        perror("vecA malloc");
        exit(EXIT_FAILURE);
    }
    if((vecB = (double *)malloc(sizeof(double)*n)) == NULL) {
        perror("vecB malloc");
        exit(EXIT_FAILURE);
    }

    if((vecC = (double *)malloc(sizeof(double)*n)) == NULL) {
        perror("vecC malloc");
        exit(EXIT_FAILURE);
    }

    hipMalloc((void**)&d_vecA,sizeof(double) * n);
    hipMalloc((void**)&d_vecB,sizeof(double) * n);
    hipMalloc((void**)&d_vecC,sizeof(double) * n);

    // Initialize vectors
    for(i=0;i<n;i++) {
        vecA[i] = 1.0 + i;
        vecB[i] = pow(vecA[i],2);
    }

    hipMemcpy(d_vecA,vecA,sizeof(double) * n,hipMemcpyHostToDevice);
    hipMemcpy(d_vecB,vecB,sizeof(double) * n,hipMemcpyHostToDevice);

    vector_add<<<blocks,threads>>>(d_vecA,d_vecB,d_vecC,n);
    cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess) {
        fprintf(stderr,"CUDA error: %s\n",hipGetErrorString(cuda_error));
        exit(EXIT_FAILURE);
    }

    hipMemcpy(vecC,d_vecC,sizeof(double) * n,hipMemcpyDeviceToHost);

    for(i=0;i<n;i++) {
        sum += vecC[i];
    }

    printf("Sum: %18.16f\n", sum);

    free(vecA);
    free(vecB);
    free(vecC);
    hipFree(d_vecA);
    hipFree(d_vecB);
    hipFree(d_vecC);

    exit(EXIT_SUCCESS);
}

void usage() {
    fprintf(stderr,"vector-add -n <vector-size> -t <threads-per-block>\n");
    fprintf(stderr,"Initializes two vectors, adds them\n");
}

__global__ void vector_add(double *vecA,double *vecB,double *vecC,const unsigned long int n) {
    unsigned long int i;

    for(i=threadIdx.x;i<n;i+=blockDim.x) {
        vecC[i] = vecA[i] + vecB[i];
    }
}
